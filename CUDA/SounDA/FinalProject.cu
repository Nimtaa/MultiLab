#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <stdio.h>
#include <stdlib.h> 
#include <fstream>
#include <iostream>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h> 
#include <algorithm> 
#include <vector>
#include <iostream>
#include <string>
#include <cmath>
#include <string>
#include <string>
#include <iostream>
#include <Windows.h>
#include <vector>
#include <filesystem>
#include <algorithm>
#include <iterator>
#include <omp.h>
#include<limits>
#include <dirent.h>


#define BATCH 1
#define CHUNK_SIZE 2000
#define NUM_THREAD 4

#define MIN_FLOAT 0.000000
#define MAX_FLOAT 99999999.9

typedef std::vector<std::string> stringVectType;

using namespace std;


__global__ void similarityKernel(hipfftComplex ** music_data, hipfftComplex ** sample_data, float *out) {
	float dis_end[5];
	__shared__ float distance[1024];
	hipfftComplex inner_sum;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	//each thread calculates the diffrence between window of song and sample.
	for (int j = 0; j < 445; j++) {
		for (int k = 0; k < CHUNK_SIZE; k++) {
			inner_sum.x += (music_data[j + i][k].x - sample_data[j][k].x);
			inner_sum.y += (music_data[j + i][k].y - sample_data[j][k].y);
		}
		__syncthreads();

		distance[i] = (float)(inner_sum.x + inner_sum.y);
	}
		//first thread of each block
		if (threadIdx.x == 0) {
			float min = MAX_FLOAT;
			for (int i = 0; i < 1024; i++) {
				if (distance[i] < min) min = distance[i];
			}
			dis_end[blockIdx.x] = min;
		}

		for(int j=0;j<5;j++){
			out[j] = dis_end[j];
		}
	
}


// Helper function to read the specific directory
stringVectType files_of_directory(string nameOfDirectory)
{
	DIR *dir;
	struct dirent *ent;
	stringVectType files;
	if ((dir = opendir(nameOfDirectory.c_str())) != NULL) {
		/* print all the files and directories within directory */
		while ((ent = readdir(dir)) != NULL) {
			//printf("%s\n", ent->d_name);
			files.push_back(ent->d_name);
		}
		closedir(dir);
	}
	else {
		/* could not open directory */
		//perror("");
		cout << "could not open directory" << endl;
		//return EXIT_FAILURE;
	}
	return files;
}

hipfftComplex **  read_music_file(string address_of_music, bool isOriginal) {

	int temp;
	int row_numbers;

	if (isOriginal) {
		row_numbers = 5000;
	}
	else {
		row_numbers = 445;
	}

	hipfftComplex** output = new hipfftComplex *[row_numbers];
	for (int i = 0; i < row_numbers; ++i)
		output[i] = new hipfftComplex[CHUNK_SIZE];

	ifstream input_music;
	input_music.open(address_of_music);

	if (!input_music) {
		cout << "A problem has occured during reading the file";
		cout << "problem with this address: " << address_of_music << endl;
		exit(1);
	}

	int i = 0;
	int j = 0;

	while (i < row_numbers)
	{

		if (input_music >> temp) {
			output[i][j].x = temp;
			output[i][j].y = 0;
		}
		else {
			output[i][j].x = 0;
			output[i][j].y = 0;
		}
		j++;
		if (j == CHUNK_SIZE) {
			j = 0;
			i++;
		}
	}
	return output;
}

float similarity_calculation(hipfftComplex ** music_data, hipfftComplex ** sample_data) {
	float* out;
	float* out_host;
	hipfftComplex** music_data_dev;
	hipfftComplex** sample_data_dev;
	hipError_t cudaStatus;
	// GPU memory allocation
	cudaStatus = hipMalloc((void **)&music_data_dev, sizeof(hipfftComplex)*CHUNK_SIZE * 5000);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		cout << "problem with hipMalloc!" << endl;
		exit(1);
	}
	cudaStatus = hipMalloc((void **)&sample_data_dev, sizeof(hipfftComplex)*CHUNK_SIZE * 440);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		exit(1);
	}
	
	cudaStatus = hipMalloc((void **)&out, 5 *sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		exit(1);
	}
	// Copy from host to device
	cudaStatus = hipMemcpy(sample_data_dev, sample_data, CHUNK_SIZE * 440 * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		exit(1);
	}
	// Copy from host to device
	cudaStatus = hipMemcpy(music_data_dev, music_data, CHUNK_SIZE * 5000 * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		exit(1);
	}
	
	//Kernel execution
	similarityKernel <<<5, 1024>>> (music_data_dev, sample_data_dev, out);

	// cout<<"Kernel Launched!"<<endl;
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(1);
	}
	// Copy from device to host
	cudaStatus = hipMemcpy(out_host, out, 5 * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		exit(1);
	}

	hipFree(music_data_dev);
	hipFree(sample_data_dev);
	hipFree(out);

	float min = MAX_FLOAT;
	for (int i = 0;i < 5;i++) {
		if (out_host[i] < min)
			min = out_host[i];
	}
	return min;
}

hipfftComplex ** get_fourier_from_input(hipfftComplex ** song, bool isOriginal) {

	int music_size;

	if (isOriginal) {
		music_size = 5000;
	}
	else {
		music_size = 445;
	}
	hipfftHandle Plan;
#pragma omp parallel for 
	for (int i = 0; i < music_size; i++) {

		hipfftComplex * device_music;

		// GPU memory allocation
		hipMalloc((void **)&device_music, sizeof(hipfftComplex)*CHUNK_SIZE);

		// Copy from host to device
		hipMemcpy(device_music, song[i], CHUNK_SIZE * sizeof(hipfftComplex), hipMemcpyHostToDevice);

		// Executing the FFT
		hipfftPlan1d(&Plan, CHUNK_SIZE, HIPFFT_C2C, BATCH);
		hipfftExecC2C(Plan, device_music, device_music, HIPFFT_FORWARD);

		// Waits for the kernel to finish, and returns any errors encountered during the execution.
		hipDeviceSynchronize();

		// Copy results from device to host
		hipMemcpy(song[i], device_music, CHUNK_SIZE * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

		hipFree(device_music);
	}
	hipfftDestroy(Plan);
	return song;
}

int main(int argc, char* argv[])
{
	// for (int i = 0;i<argc;i++) {
	// 	cout << argv[i] << endl;
	// }

	string sample_directory;
	string music_directory;

	stringVectType sample_files;
	stringVectType song_files;

	sample_directory = argv[2];
	music_directory = argv[1];

	sample_files = files_of_directory(".\\" + sample_directory);
	song_files = files_of_directory(".\\" + music_directory);

	for (int i = 2; i < sample_files.size(); i++) {
		
		hipfftComplex** sample_result = read_music_file(".\\"+ sample_directory +"\\" + sample_files[i], 0);
		sample_result = get_fourier_from_input(sample_result, 0);
		float min = 1000;
		int counter = 0;
		for (int j = 2; j < song_files.size();j++) {
			hipfftComplex** music_result = read_music_file(".\\" + music_directory +"\\" + song_files[i], 1);
			music_result = get_fourier_from_input(music_result, 1);
			float distance = similarity_calculation(music_result, sample_result);
			if (distance < min) {
				min = distance;
				counter++;
			}
		}
		if (counter == 0) {
			cout << "S" << i - 1 << ".wav>>>" << "NotFound" << endl;
		}else{
			cout << "S" << i - 1 << ".wav>>>" << counter << ".txt"<<endl;
		}
	}
	getchar();
	return 0;
}
